#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <math.h>
#include "vector.h"
#include "config.h"
#include <stdio.h>

__global__ void allocate_accels(vector3** d_accels, vector3* d_values) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < NUMENTITIES)
		d_accels[i]=&d_values[i*NUMENTITIES];
}

__global__ void compute_accels(vector3** d_accels, vector3* d_hVel, vector3* d_hPos, double* d_mass) {
	int i,j,k;
	for (i=0;i<NUMENTITIES;i++){
		for (j=0;j<NUMENTITIES;j++){
			if (i==j) {
				FILL_VECTOR(d_accels[i][j],0,0,0);
			}
			else{
				vector3 distance;
				for (k=0;k<3;k++) distance[k]=d_hPos[i][k]-d_hPos[j][k];
				double magnitude_sq=distance[0]*distance[0]+distance[1]*distance[1]+distance[2]*distance[2];
				double magnitude=sqrt(magnitude_sq);
				double accelmag=-1*GRAV_CONSTANT*d_mass[j]/magnitude_sq;
				FILL_VECTOR(d_accels[i][j],accelmag*distance[0]/magnitude,accelmag*distance[1]/magnitude,accelmag*distance[2]/magnitude);
			}
		}
	}
	//sum up the rows of our matrix to get effect on each entity, then update velocity and position.
	for (i=0;i<NUMENTITIES;i++){
		vector3 accel_sum={0,0,0};
		for (j=0;j<NUMENTITIES;j++){
			for (k=0;k<3;k++)
				accel_sum[k]+=d_accels[i][j][k];
		}
		//compute the new velocity based on the acceleration and time interval
		//compute the new position based on the velocity and time interval
		for (k=0;k<3;k++){
			d_hVel[i][k]+=accel_sum[k]*INTERVAL;
			d_hPos[i][k]+=d_hVel[i][k]*INTERVAL;
		}
	}
}

__global__ void print_accels(vector3** d_accels) {
	for (int i = 0; i < NUMENTITIES; i++) {
		for (int j = 0; j < NUMENTITIES; j++) {
			printf("%d   ", d_accels[i][j][0]);
		}
		printf("\n");
	}
}

//compute: Updates the positions and locations of the objects in the system based on gravity.
//Parameters: None
//Returns: None
//Side Effect: Modifies the hPos and hVel arrays with the new positions and accelerations after 1 INTERVAL
void compute(){
	//make an acceleration matrix which is NUMENTITIES squared in size;
	vector3* d_values;
	vector3** d_accels;
	hipMalloc(&d_values, sizeof(vector3)*NUMENTITIES*NUMENTITIES);
	hipMalloc(&d_accels, sizeof(vector3*)*NUMENTITIES);
	dim3 threadsPerBlock(16,16);
	int numBlocks = (NUMENTITIES + threadsPerBlock.x - 1) / threadsPerBlock.x;

	allocate_accels<<<numBlocks,threadsPerBlock>>>(d_accels, d_values);
	//compute_accels<<<1, 1>>>(d_accels, d_hVel, d_hPos, d_mass);
	//print_accels<<<1, 1>>>(d_accels);
	hipMemcpy(hVel, d_hVel, sizeof(vector3) * NUMENTITIES, hipMemcpyDeviceToHost);
	hipMemcpy(hPos, d_hPos, sizeof(vector3) * NUMENTITIES, hipMemcpyDeviceToHost);
	/*
	//first compute the pairwise accelerations.  Effect is on the first argument.
	for (i=0;i<NUMENTITIES;i++){
		for (j=0;j<NUMENTITIES;j++){
			if (i==j) {
				FILL_VECTOR(accels[i][j],0,0,0);
			}
			else{
				vector3 distance;
				for (k=0;k<3;k++) distance[k]=hPos[i][k]-hPos[j][k];
				double magnitude_sq=distance[0]*distance[0]+distance[1]*distance[1]+distance[2]*distance[2];
				double magnitude=sqrt(magnitude_sq);
				double accelmag=-1*GRAV_CONSTANT*mass[j]/magnitude_sq;
				FILL_VECTOR(accels[i][j],accelmag*distance[0]/magnitude,accelmag*distance[1]/magnitude,accelmag*distance[2]/magnitude);
			}
		}
	}
	//sum up the rows of our matrix to get effect on each entity, then update velocity and position.
	for (i=0;i<NUMENTITIES;i++){
		vector3 accel_sum={0,0,0};
		for (j=0;j<NUMENTITIES;j++){
			for (k=0;k<3;k++)
				accel_sum[k]+=accels[i][j][k];
		}
		//compute the new velocity based on the acceleration and time interval
		//compute the new position based on the velocity and time interval
		for (k=0;k<3;k++){
			hVel[i][k]+=accel_sum[k]*INTERVAL;
			hPos[i][k]+=hVel[i][k]*INTERVAL;
		}
	}

	/* for reduction
	for (i=0;i<NUMENTITIES;i++) {
		for (k=0;k<3;k++){ 
			hVel[i][k]+=accel_sum[k]*INTERVAL;
			hPos[i][k]+=hVel[i][k]*INTERVAL;
		}
	}
	*/
	hipFree(d_values);
	hipFree(d_accels);
}

#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <math.h>
#include "vector.h"
#include "config.h"
#include "planets.h"
#include "compute.h"
// represents the objects in the system.  Global variables
vector3 *hVel, *d_hVel;
vector3 *hPos, *d_hPos;
double *mass, *d_mass;
vector3** d_accels;
vector3* d_values;

//initHostMemory: Create storage for numObjects entities in our system
//Parameters: numObjects: number of objects to allocate
//Returns: None
//Side Effects: Allocates memory in the hVel, hPos, and mass global variables
void initHostMemory(int numObjects)
{
	hVel = (vector3 *)malloc(sizeof(vector3) * numObjects);
	hPos = (vector3 *)malloc(sizeof(vector3) * numObjects);
	mass = (double *)malloc(sizeof(double) * numObjects);
}

void initDeviceMemory(int numObjects)
{
	hipMalloc(&d_values, sizeof(vector3)*numObjects*numObjects);
	hipMalloc(&d_accels, sizeof(vector3*)*numObjects);
	
	hipMalloc(&d_mass, sizeof(double) * numObjects);
}

void copyToDevice(int numObjects) {
	hipMemcpy(d_hVel, hVel, sizeof(vector3) * numObjects, hipMemcpyHostToDevice);
	hipMemcpy(d_hPos, hPos, sizeof(vector3) * numObjects, hipMemcpyHostToDevice);
	hipMemcpy(d_mass, mass, sizeof(double) * numObjects, hipMemcpyHostToDevice);
}

//freeHostMemory: Free storage allocated by a previous call to initHostMemory
//Parameters: None
//Returns: None
//Side Effects: Frees the memory allocated to global variables hVel, hPos, and mass.
void freeHostMemory()
{
	free(hVel);
	free(hPos);
	free(mass);
}

void freeDeviceMemory()
{
	hipFree(d_accels);
	hipFree(d_values);
	hipFree(d_hVel);
	hipFree(d_hPos);
	hipFree(d_mass);
}

//planetFill: Fill the first NUMPLANETS+1 entries of the entity arrays with an estimation
//				of our solar system (Sun+NUMPLANETS)
//Parameters: None
//Returns: None
//Fills the first 8 entries of our system with an estimation of the sun plus our 8 planets.
void planetFill(){
	int i,j;
	double data[][7]={SUN,MERCURY,VENUS,EARTH,MARS,JUPITER,SATURN,URANUS,NEPTUNE};
	for (i=0;i<=NUMPLANETS;i++){
		for (j=0;j<3;j++){
			hPos[i][j]=data[i][j];
			hVel[i][j]=data[i][j+3];
		}
		mass[i]=data[i][6];
	}
}

//randomFill: FIll the rest of the objects in the system randomly starting at some entry in the list
//Parameters: 	start: The index of the first open entry in our system (after planetFill).
//				count: The number of random objects to put into our system
//Returns: None
//Side Effects: Fills count entries in our system starting at index start (0 based)
void randomFill(int start, int count)
{
	int i, j = start;
	for (i = start; i < start + count; i++)
	{
		for (j = 0; j < 3; j++)
		{
			hVel[i][j] = (double)rand() / RAND_MAX * MAX_DISTANCE * 2 - MAX_DISTANCE;
			hPos[i][j] = (double)rand() / RAND_MAX * MAX_VELOCITY * 2 - MAX_VELOCITY;
			mass[i] = (double)rand() / RAND_MAX * MAX_MASS;
		}
	}
}

//printSystem: Prints out the entire system to the supplied file
//Parameters: 	handle: A handle to an open file with write access to prnt the data to
//Returns: 		none
//Side Effects: Modifies the file handle by writing to it.
void printSystem(FILE* handle){
	int i,j;
	for (i=0;i<NUMENTITIES;i++){
		fprintf(handle,"pos=(");
		for (j=0;j<3;j++){
			fprintf(handle,"%lf,",hPos[i][j]);
		}
		fprintf(handle,"),v=(");
		for (j=0;j<3;j++){
			fprintf(handle,"%lf,",hVel[i][j]);
		}
		fprintf(handle,"),m=%lf\n",mass[i]);
	}
}

__global__ void init_accels(vector3** d_accels, vector3* d_values) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < NUMENTITIES)
		d_accels[i]=&d_values[i*NUMENTITIES];
}

int main(int argc, char **argv)
{
	clock_t t0=clock();
	int t_now;
	//srand(time(NULL));
	srand(1234);
	initHostMemory(NUMENTITIES);
	initDeviceMemory(NUMENTITIES);
	
	planetFill();
	randomFill(NUMPLANETS + 1, NUMASTEROIDS);

	copyToDevice(NUMENTITIES);
	
	//now we have a system.
#ifdef DEBUG
    FILE* handle;
    handle = fopen("parallel.txt", "w+");
	printSystem(handle);
#endif

	init_accels<<<ceil((NUMENTITIES + 256-1) / 256),256>>>(d_accels, d_values); // should launch a little over NUMENTITIES threads
	hipDeviceSynchronize();
	for (t_now=0;t_now<DURATION;t_now+=INTERVAL){
		compute();
	}
	

	clock_t t1=clock()-t0;

#ifdef DEBUG
	printSystem(handle);
#endif

	printf("This took a total time of %f seconds\n",(double)t1/CLOCKS_PER_SEC);

	freeHostMemory();
	freeDeviceMemory();
}
